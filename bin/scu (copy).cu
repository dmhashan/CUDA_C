#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(int *C, int *A, int *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    int Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ int As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void constantInit(int *data, int size, int val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(int argc, char **argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(int) * size_A;
    int *h_A = (int *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(int) * size_B;
    int *h_B = (int *)malloc(mem_size_B);

    // Initialize host memory
    const int valB = 1;
    constantInit(h_A, size_A, 1);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    int *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(int);
    int *h_C = (int *) malloc(mem_size_C);

   	hipMalloc((void **) &d_A, mem_size_A);

	hipMalloc((void **) &d_B, mem_size_B);

    	hipMalloc((void **) &d_C, mem_size_C);

    	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    hipDeviceSynchronize();

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start);
        if (block_size == 16)
        {
            matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
hipEventRecord(stop);

hipEventSynchronize(stop);
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start, stop);
fprintf(stdout, "%f", milliseconds);    
hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

return 0;
}


int main(int argc, char **argv)
{
    if(argc != 3)
	exit(2);
    int N = atoi(argv[2]);
    int block_size = atoi(argv[1]);
    
    dim3 dimsA(2*N*block_size, 2*N*block_size, 1);
    dim3 dimsB(2*N*block_size, 2*N*block_size, 1);
    
    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}
